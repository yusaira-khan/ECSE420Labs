#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

__global__ void convolve(unsigned char * d_output_image, unsigned char * d_input_image){

    float w[3][3] =
    {
        {1,2,-1},
        {2,0.25,-2},
        {1,-2,-1}
        
    };
    int new_width = blockDim.x;
	int width = new_width + 2;
    int new_i = blockIdx.x;
    int new_j = threadIdx.x;
    unsigned int new_index = (4 * new_width  * new_i) + (4 * new_j);
    unsigned int old_index = (4 * (width ) * new_i) + (4 * new_j);

    
    unsigned char clamped;
    float unclamped = 0;
    for (int c = 0; c < 3; c++) {
        unclamped = 0;
        for (int ii = 0; ii < 3; ii++) {
            for (int jj = 0; jj < 3; jj++) {
                unclamped += d_input_image[old_index + (4 * width * ii) + (4 * jj) + c] * w[ii][jj];
            }
        }
        if (unclamped < 0) clamped = 0;
        else if (unclamped > 255) clamped = 255;
        else clamped = (unsigned char) unclamped;

        d_output_image[new_index + c] = clamped; // R  
    }
    d_output_image[new_index + 3] = d_input_image[old_index + (4 * (width)) + 4  + 3]; // A

}

int main(int argc, char **argv){
	char *input_filename = argv[1];
	char *output_filename = argv[2];
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	unsigned char *d_in;
	unsigned char *d_out;
	int img_size, new_img_size;
//	const int MAX_THREADS = 1024;
	int numThreads, numBlocks;
	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error)
		printf("error %u: %s\n", error, lodepng_error_text(error));
	img_size = width * height * sizeof(unsigned char) * 4;
	new_img_size = (width-2) * (height-2) * sizeof(unsigned char) * 4;
	new_image = (unsigned char *)malloc(new_img_size);

	numThreads = (width-2);
	numBlocks = (height-2);

	hipMalloc(&d_in, img_size);
	hipMalloc(&d_out, new_img_size);

	hipMemcpy(d_in, image, img_size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(numThreads, 1, 1);		
	dim3 dimGrid(numBlocks, 1, 1);

	convolve<<<dimGrid, dimBlock>>>(d_out, d_in);

	hipMemcpy(new_image, d_out, new_img_size, hipMemcpyDeviceToHost);
	
	lodepng_encode32_file(output_filename, new_image, width-2, height-2);

	hipFree(d_in);
	hipFree(d_out);
	free(image);
	free(new_image);
}
