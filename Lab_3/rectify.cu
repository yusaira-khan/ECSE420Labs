#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

__global__ void rectify(unsigned char *d_out, unsigned char *d_in){
	
	int idx = threadIdx.x;
	int block = blockIdx.x;
	int width = blockDim.x;
	int index = 4*(block*width + idx);
	d_out[index + 0] = d_in[index + 0] < 127 ? 127 : d_in[index + 0];
	d_out[index + 1] = d_in[index + 1] < 127 ? 127 : d_in[index + 1];
	d_out[index + 2] = d_in[index + 2] < 127 ? 127 : d_in[index + 2];
	d_out[index + 3] = 225;
}

int main(int argc, char ** argv){
	char *input_filename = argv[1];
	char *output_filename = argv[2];

	unsigned error;
	unsigned char *image, *new_image;
	unsigned char *d_in, *d_out;
	unsigned width, height;
	int img_size, img_bytes;
	int MAX_THREADS = 1024;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error)
		printf("error %u: %s\n", error, lodepng_error_text(error));
	img_size = width * height;
	img_bytes = img_size*4;
	new_image = (unsigned char *)malloc(img_bytes *sizeof(unsigned char));
	hipMalloc(&d_in, img_bytes);
	hipMalloc(&d_out, img_bytes);

	hipMemcpy(d_in, image, img_bytes, hipMemcpyHostToDevice);

//	dim3 dimBlock();
//	dim3 dimGrid();

	rectify<<<img_size/MAX_THREADS, MAX_THREADS>>>(d_out, d_in);
//	rectify<<<MAX_THREADS, img_size/MAX_THREADS>>>(d_out, d_in);

	int remainder = img_size%MAX_THREADS;

	hipMemcpy(new_image, d_out, img_bytes, hipMemcpyDeviceToHost);

	for(int idx = img_size - remainder; idx< img_size; idx++){
		new_image[4*idx+0] = image[4*idx+0] < 127 ? 127 : image[4*idx+0];
		new_image[4*idx+1] = image[4*idx+1] < 127 ? 127 : image[4*idx+1];
		new_image[4*idx+2] = image[4*idx+2] < 127 ? 127 : image[4*idx+2];
		new_image[4*idx+3] = 225;
	}
	
	lodepng_encode32_file(output_filename, new_image, width, height);
	free(image);
	free(new_image);
}
