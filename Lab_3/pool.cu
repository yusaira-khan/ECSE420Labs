#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

__global__ void pool(unsigned char *d_out, unsigned char *d_in){
	int idx = threadIdx.x;
	int block = blockIdx.x;
	int width = blockDim.x;

	int index = 4*(width*block + idx);
	int indexSlice0 = 4*(width*2*block*2 + idx*2);
	int indexSlice1 = 4*(width*2*block*2 + idx*2 + 1);
	int indexSlice2 = 4*(width*2*(block*2 + 1) + idx*2);
	int indexSlice3 = 4*(width*2*(block*2 + 1) + idx*2 + 1);

	
	unsigned char sliceR[4];
	unsigned char sliceG[4];
	unsigned char sliceB[4];
	unsigned char sliceA[4];
	
	sliceR[0] = d_in[indexSlice0];
	sliceR[1] = d_in[indexSlice1];
	sliceR[2] = d_in[indexSlice2];
	sliceR[3] = d_in[indexSlice3];
	
	sliceG[0] = d_in[indexSlice0 + 1];
	sliceG[1] = d_in[indexSlice1 + 1];
	sliceG[2] = d_in[indexSlice2 + 1];
	sliceG[3] = d_in[indexSlice3 + 1];
	
	sliceB[0] = d_in[indexSlice0 + 2];
	sliceB[1] = d_in[indexSlice1 + 2];
	sliceB[2] = d_in[indexSlice2 + 2];
	sliceB[3] = d_in[indexSlice3 + 2];

	unsigned char max = 0;
	for(unsigned j=0; j<4; j++){
		if (sliceR[j] > max)
			max = sliceR[j];
	}
	d_out[index] = max;
	
	max = 0;
	for(unsigned j=0; j<4; j++){
		if (sliceG[j] > max)
			max = sliceG[j];
	}
	d_out[index + 1] = max;
	
	max = 0;
	for(unsigned j=0; j<4; j++){
		if (sliceB[j] > max)
			max = sliceB[j];
	}
	d_out[index + 2] = max;
	
	d_out[index + 3] = d_in[indexSlice0 + 3];;
	
}

int main(int argc, char **argv){
	char *input_filename = argv[1];
	char *output_filename = argv[2];
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	unsigned char *d_in;
	unsigned char *d_out;
	int img_size, new_img_size;
//	const int MAX_THREADS = 1024;
	int numThreads, numBlocks;
	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error)
		printf("error %u: %s\n", error, lodepng_error_text(error));
	img_size = width * height * sizeof(unsigned char) * 4;
	new_img_size = width * height * sizeof(unsigned char);
	new_image = (unsigned char *)malloc(new_img_size);

	numThreads = width/2;
	numBlocks = height/2;

	hipMalloc(&d_in, img_size);
	hipMalloc(&d_out, new_img_size);

	hipMemcpy(d_in, image, img_size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(numThreads, 1, 1);		
	dim3 dimGrid(numBlocks, 1, 1);

	pool<<<dimGrid, dimBlock>>>(d_out, d_in);

	hipMemcpy(new_image, d_out, new_img_size, hipMemcpyDeviceToHost);
	
	lodepng_encode32_file(output_filename, new_image, width/2, height/2);

	hipFree(d_in);
	hipFree(d_out);
	free(image);
	free(new_image);
}
