#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

//2) how many threads do you need? #pixels/4 -> as many threads as size of d_out
//3) #blocks = #threads/512
//4) d_in is the whole array so need to figure out how to extract indices
// or maybe pass the indices to pool method?
//5) d_out is N/4 while d_in is N size
//6) need 3D blocks? i, j for pixel grid and k for color?
//7) how do you define size of block? want # of idx equal to width, # of idy equal to height and # of idz equal to 4: dim3
//8) how do you pass the width to the gpu since you need that to compute the d_in and d_out indices
//9) have to consider number of blocks if img size>512


__global__ void pool(unsigned char *d_out, unsigned char *d_in){
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	int idz = threadIdx.z;
	int width = blockDim.x;
	int height = blockDim.y;
	int block = blockIdx.x;
	int size = sizeof(d_out)/sizeof(d_out[0]);
	int index = block*width*height*4 + 4*(idx + width*idy) + idz;
	printf("size is %d and index is %d\n", size, index);
//	printf("blockDim.x is %d, blockDim.y is %d, blockDim.z is %d, x is %d, y is %d, z is %d\n", blockDim.x, blockDim.y, blockDim.z, idx, idy, idz);
/*	if ((idx%2 !=0) || (idy%2 != 0)){

	} 
	else*/
	if(index < size) { 
	if(idz < 3){
		unsigned char slice[4];
		slice[0] = d_in[block*width*2*height*2 + 4*(idx*2 + width*2*idy*2) + idz];
		slice[1] = d_in[block*width*2*height*2 + 4*(idx*2 + width*2*(idy*2 +1)) + idz];
		slice[2] = d_in[block*width*2*height*2 + 4*(idx*2 + 1 + width*2*idy*2) + idz];
		slice[3] = d_in[block*width*2*height*2 + 4*(idx*2 + 1 + width*2*(idy*2 + 1)) + idz];
//		printf("x is %d, y is %d, z is %d and slice is %d, %d, %d, %d\n", idx, idy, idz, slice[0], slice[1], slice[2], slice[3]);
/*		for(unsigned i=0; i<4; i++){
			slice[i] = i;
		}	
*/		unsigned char max = 0;
		for(unsigned i=0; i<4; i++){
			if (slice[i] > max)
				max = slice[i];
		}
		d_out[index] = max;
//		printf("x is %d, y is %d, z is %d, slice is %d, %d, %d, %d and d_out is %d\n", idx, idy, idz, slice[0], slice[1], slice[2], slice[3], d_out[idx/2 + blockDim.x*idy/4 +idz]);
	}
	else {
		d_out[index] = d_in[block*width*2*height*2 + 4*(idx*2 + width*2*idy*2) + 3];
//		printf("x is %d, y is %d, z is %d, and d_out is %d\n", idx, idy, idz, d_out[idx/2 + blockDim.x*idy/4 +idz]);
	}
	}
}

int main(int argc, char **argv){
//	char *input_filename = argv[1];
//	char *output_filename = argv[2];
//	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	unsigned char *d_in;
	unsigned char *d_out;
	int img_size, new_img_size;
	const int MAX_THREADS = 60;
	int blockW, blockH, numBlocks;
/*	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error)
		printf("error %u: %s\n", error, lodepng_error_text(error));
	img_size = width * height * sizeof(unsigned char) * 4;
	new_img_size = width * height * sizeof(unsigned char)
	new_image = malloc(new_img_size);
*/
	img_size = 64*4;
	new_img_size = img_size/4;
	image = (unsigned char *)malloc(img_size*sizeof(unsigned char));
	new_image = (unsigned char *)malloc(new_img_size*sizeof(unsigned char));
	width = 8;
	height = 8;

	if(new_img_size > MAX_THREADS){
		if (width < MAX_THREADS/2){
			blockW = width/2;
			blockH = MAX_THREADS/(blockW * 4);
			
		} else {
			blockW = MAX_THREADS/4;
			blockH = 1;
		}
		numBlocks = (new_img_size/(blockW * blockH * 4));
		if ( new_img_size%(blockW * blockH * 4) != 0) {	
			numBlocks++;
		}	
	} else {
		blockW = width/2;
		blockH = height/2;
		numBlocks = 1;
	}

	for(unsigned i=0; i<img_size; i++){
		image[i] = i;
	}
	hipMalloc(&d_in, img_size);
	hipMalloc(&d_out, new_img_size);

	hipMemcpy(d_in, image, img_size, hipMemcpyHostToDevice);
	
	dim3 dimBlock(blockW, blockH, 4);		
	dim3 dimGrid(numBlocks, 1, 1);
	
	pool<<<dimGrid, dimBlock>>>(d_out, d_in);

	hipMemcpy(new_image, d_out, new_img_size, hipMemcpyDeviceToHost);
	
	for(int i=0; i<img_size; i++){
		printf("%u; ", image[i]);
	}
	for(int i=0; i<16; i++){
		printf("\ni is %d and new image is %u\n", i, new_image[i]);
	}

	free(image);
	free(new_image);
}
