
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void update(float * d_u, float * d_u1, float * d_u2){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	float rho = 0.5;
	float eta = 0.0002;
	float g = 0.75;
	int size = 512;
	
	//Internal
	if(idx % size != 0 && idx % size != (size-1) && idx > size-1 && idx < size*size - size) { 
		d_u[idx] = (rho * (d_u1[idx - size] + d_u1[idx + size] + d_u1[idx - 1] + d_u1[idx + 1] - 4 * d_u1[idx]) + 2 * d_u1[idx] - (1 - eta) * d_u2[idx]) / (1 + eta);
	} 
	//Edges
	else if(idx % size == 0 && idx != 0 && idx != size * size - size) {
		d_u[idx] = g * ((rho * (d_u1[idx + 1 - size] + d_u1[idx + 1 + size] + d_u1[idx] + d_u1[idx + 2] - 4 * d_u1[idx + 1]) + 2 * d_u1[idx + 1] - (1 - eta) * d_u2[idx + 1]) / (1 + eta));
	} else if(idx % size == size-1 && idx != size - 1 && idx != size * size - 1 ) { 
		d_u[idx] = g * ((rho * (d_u1[idx - 1 - size] + d_u1[idx - 1 + size] + d_u1[idx - 2] + d_u1[idx] - 4 * d_u1[idx - 1]) + 2 * d_u1[idx - 1] - (1 - eta) * d_u2[idx - 1]) / (1 + eta));
	} else if(idx < size - 1 && idx > 0) { //edge
		d_u[idx] = g * ((rho * (d_u1[idx] + d_u1[idx + 2 * size] + d_u1[idx + size - 1] + d_u1[idx + size + 1] - 4 * d_u1[idx + size]) + 2 * d_u1[idx + size] - (1 - eta) * d_u2[idx + size]) / (1 + eta));
	} else if(idx < size * size - 1 && idx > size*size - size) { 
		d_u[idx] = g * ((rho * (d_u1[idx -  2 * size] + d_u1[idx] + d_u1[idx - size - 1] + d_u1[idx - size + 1] - 4 * d_u1[idx - size]) + 2 * d_u1[idx - size] - (1 - eta) * d_u2[idx - size]) / (1 + eta));
	} 

	//Corners
	else if(idx == 0) { 
		d_u[idx] = g * g * ((rho * (d_u1[idx + 1] + d_u1[idx + 2 * size + 1 ] + d_u1[idx + size] + d_u1[idx + size + 2] - 4 * d_u1[idx + size + 1]) + 2 * d_u1[idx + size + 1] - (1 - eta) * d_u2[idx + size + 1]) / (1 + eta));
	} else if ( idx == size - 1) {
		d_u[idx] = g * g * ((rho * (d_u1[idx  - 1] + d_u1[idx + 2 * size - 1] + d_u1[idx + size - 2] + d_u1[idx + size] - 4 * d_u1[idx + size - 1]) + 2 * d_u1[idx + size - 1] - (1 - eta) * d_u2[idx + size - 1]) / (1 + eta));
	} else if ( idx == size * size - 1 ) { 
		d_u[idx] = g * g * ((rho * (d_u1[idx - 2 * size - 1] + d_u1[idx - 1] + d_u1[idx - size - 2] + d_u1[idx - size] - 4 * d_u1[idx - size - 1]) + 2 * d_u1[idx - size - 1] - (1 - eta) * d_u2[idx - size - 1]) / (1 + eta));
	} else if ( idx == size * size - size ) { 
		d_u[idx] = g * g * ((rho * (d_u1[idx - 2 * size + 1] + d_u1[idx  + 1] + d_u1[idx - size] + d_u1[idx - size + 2] - 4 * d_u1[idx - size + 1]) + 2 * d_u1[idx - size + 1] - (1 - eta) * d_u2[idx - size + 1]) / (1 + eta));
	}
	
	if(idx == (size*size/2 + size/2)) {
		printf("%f, \n", d_u[idx]);
	}
}



int main(int argc, char ** argv) {
	int iterations = atoi(argv[1]);
	const int size = 512 * 512;
	
	float *h_u = (float *)malloc(size * sizeof(float));
	float *h_u1 = (float *)malloc(size * sizeof(float));
	float *h_u2 = (float *)malloc(size * sizeof(float));
	for (int j = 0; j < size; j++) {
		h_u[j] = 0;
		h_u1[j] = 0;
		h_u2[j] = 0;
	}

	h_u1[size / 2 + 256] = 1.0;
	float *d_u;
	float *d_u1;
	float *d_u2;

	hipMalloc(&d_u, size * sizeof(float));
	hipMalloc(&d_u1, size * sizeof(float));
	hipMalloc(&d_u2, size * sizeof(float));

	hipMemcpy(d_u, h_u, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u1, h_u1, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u2, h_u2, size * sizeof(float), hipMemcpyHostToDevice);

	free(h_u);
	free(h_u1);
	free(h_u2);

	for (int i = 0; i < iterations; i++) {
		update<<<512, 512>>>(d_u, d_u1, d_u2);
		float* temp = d_u2;
    	d_u2 = d_u1;
    	d_u1 = d_u;
  		d_u = temp;
	}

	hipFree(d_u);
	hipFree(d_u1);
	hipFree(d_u2);

	return 0;
}

