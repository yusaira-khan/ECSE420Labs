
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void update(float *d_out, float *d_in) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	float rho = 0.5;
	float eta = 0.0002;
	float G = 0.75;
	int size = 4;
	
	//update center
	if((idx/size != 0) && (idx/size != 3) && (idx%size != 0) && (idx%size != 3)) {
		d_out[idx] = (rho*(d_in[idx-1] + d_in[idx+1] + d_in[idx-size] +d_in[idx+size] - 4*d_in[idx]) + 2*d_in[idx] - (1-eta)*d_in[idx+size*size])/(1+eta);
	}
	
	//update sides
	else if((idx/size == 0) && (idx != 0) && (idx != (size -1))){
		d_out[idx] = G*(rho*(d_in[idx-1+size] + d_in[idx+1+size] + d_in[idx-size+size] +d_in[idx+size+size] - 4*d_in[idx+size]) + 2*d_in[idx+size] - (1-eta)*d_in[idx+size*size+size])/(1+eta);
	}
	else if((idx/size == 3) && (idx != (size-1)*size) && (idx != (size*size - 1))){
		d_out[idx] = G*(rho*(d_in[idx-1-size] + d_in[idx+1-size] + d_in[idx-size-size] +d_in[idx+size-size] - 4*d_in[idx-size]) + 2*d_in[idx-size] - (1-eta)*d_in[idx+size*size-size])/(1+eta);
	}
	
	else if ((idx%size == 0) && (idx != 0) && (idx != (size-1)*size)){
		d_out[idx] = G*(rho*(d_in[idx-1+1] + d_in[idx+1+1] + d_in[idx-size+1] +d_in[idx+size+1] - 4*d_in[idx+1]) + 2*d_in[idx+1] - (1-eta)*d_in[idx+size*size+1])/(1+eta);
	}
	
	else if((idx%size == 3) && (idx != (size -1)) && (idx != (size*size - 1))){
		d_out[idx] = G*(rho*(d_in[idx-1-1] + d_in[idx+1-1] + d_in[idx-size-1] +d_in[idx+size-1] - 4*d_in[idx-1]) + 2*d_in[idx-1] - (1-eta)*d_in[idx+size*size-1])/(1+eta);
	}
	
	//update corners
	else if (idx == 0){
		d_out[idx] = G*G*(rho*(d_in[idx-1+size+1] + d_in[idx+1+size+1] + d_in[idx-size+size+1] +d_in[idx+size+size+1] - 4*d_in[idx+size+1]) + 2*d_in[idx+size+1] - (1-eta)*d_in[idx+size*size+size+1])/(1+eta);
	} else if(idx == (size - 1)){
		d_out[idx] = G*G*(rho*(d_in[idx-1+size-1] + d_in[idx+1+size-1] + d_in[idx-size+size-1] +d_in[idx+size+size-1] - 4*d_in[idx+size-1]) + 2*d_in[idx+size-1] - (1-eta)*d_in[idx+size*size+size-1])/(1+eta);
	} else if (idx == (size-1)*size){
		d_out[idx] = G*G*(rho*(d_in[idx-1-size+1] + d_in[idx+1-size+1] + d_in[idx-size-size+1] +d_in[idx+size-size+1] - 4*d_in[idx-size+1]) + 2*d_in[idx-size+1] - (1-eta)*d_in[idx+size*size-size+1])/(1+eta);
	} else if (idx == (size*size - 1)){
		d_out[idx] = G*G*(rho*(d_in[idx-1-size-1] + d_in[idx+1-size-1] + d_in[idx-size-size-1] +d_in[idx+size-size-1] - 4*d_in[idx-size-1]) + 2*d_in[idx-size-1] - (1-eta)*d_in[idx+size*size-size-1])/(1+eta);
	}
}

int main(int argc, char **argv){
	int iterations = atoi(argv[1]);
	int size = 16;
	int size_bytes = size*sizeof(float);
	float u1_2[2*size];
	float u[size];
	float *d_in, *d_out;

//initialize arrays
	for(int i=0; i<2*size; i++){
		if (i==10)
			u1_2[i] = 1;
		else
			u1_2[i] = 0;
	}
	
	for(int i=0; i<size; i++){
		u[i] = 0;
	}

	hipMalloc(&d_in, 2*size_bytes);
	hipMalloc(&d_out, size_bytes);
	
	for(int i=0; i<iterations; i++){
		hipMemcpy(d_in, u1_2, 2*size_bytes, hipMemcpyHostToDevice);

		update<<<size, 1>>>(d_out, d_in);

		hipMemcpy(u, d_out, size_bytes, hipMemcpyDeviceToHost);	
		
		for(int j=0; j<size; j++){
			u1_2[j+size] = u1_2[j];
		}

		for(int j=0; j<size; j++){
			u1_2[j] = u[j];
		}
		
		printf("%f\n", u[10]);
	}
	
	hipFree(d_in);
	hipFree(d_out);
}
