
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void updateCenter(float *d_out, float *d1_in, float *d2_in) {
	int idx = threadIdx.x;
	float rho = 0.5;
	float eta = 0.0002;
	int size = 4;
	printf("u1: ");
	for(int i=0; i<size*size; i++){
		printf("%f, ", d1_in[i]);
	}
	printf("\nu2: ");
	for(int i=0; i<size*size; i++){
		printf("%f, ", d2_in[i]);
	}
	printf("\n");
	if((idx/size != 0) && (idx/size != 3) && (idx%size != 0) && (idx%size != 3)) {
		d_out[idx] = (rho*(d1_in[idx-1] + d1_in[idx+1] + d1_in[idx-size] +d1_in[idx+size] - 4*d1_in[idx]) + 2*d1_in[idx] - (1-eta)*d2_in[idx])/(1+eta);
	} else {
		d_out[idx] = 0;
	}
	
//	printf("updating center and idx is %d, u[idx] is %f, u1[idx-1] is %f, u1[idx+1] is %f, u1[idx-size] is %f, u1[idx+size] is %f, u1[idx] is %f and u2[idx] is %f\n", idx, d_out[idx], d1_in[idx-1], d1_in[idx+1], d1_in[idx-size], d1_in[idx+size], d1_in[idx], d2_in[idx]);

}

__global__ void updateSides(float *d_out, float *d_in){
	int idx = threadIdx.x;
	int size = 4;
	float G = 0.75;

	if((idx/size == 0) && (idx != 0) && (idx != (size -1))){
		d_out[idx] = G*d_in[idx + size];
	}
	else if((idx/size == 3) && (idx != (size-1)*size) && (idx != (size*size - 1))){
		d_out[idx] = G*d_in[idx - size];
	}
	
	else if ((idx%size == 0) && (idx != 0) && (idx != (size-1)*size)){
		d_out[idx] = G*d_in[idx + 1];
	}
	
	else if((idx%size == 3) && (idx != (size -1)) && (idx != (size*size - 1))){
		d_out[idx] = G*d_in[idx - 1];
	}
	else {
		d_out[idx] = d_in[idx];
	}
	
//	printf("updating sides and idx is %d, and u[idx] is %f\n", idx, d_out[idx]);

}

__global__ void updateCorners(float *d_out, float *d_in){
	int idx = threadIdx.x;
	int size = 4;
	float G = 0.75;

	if ((idx == 0) || (idx == (size - 1))){
		d_out[idx] = G*d_in[idx + size];
	} else if ((idx == (size-1)*size) || (idx == (size*size - 1))){
		d_out[idx] = G*d_in[idx - size];
	} else {
		d_out[idx] = d_in[idx];
	}
	
//	printf("updating corners and idx is %d, and u[idx] is %f\n", idx, d_out[idx]);

}

int main(int argc, char **argv){
	int iterations = atoi(argv[1]);
	int size = 16;
	float u2[size];
	float u1[size];
	float u[size];
	float *d1_in, *d2_in, *d_in, *d_out;
/*	float *center_in, *center_out;
	float *sides_in, *sides_out;
	float *corners_in, *corners_out;
	int center_size = 7;
	int sides_size = 4;
	int corners_size = 4;
*///initialize arrays
	for(int i=0; i<size; i++){
	//	for(int j=0; j<4; j++){
			u2[i] = 0;
			if (i==10)
				u1[i] = 1;
			else
				u1[i] = 0;
	//	}
	}

/*	cudaMalloc(&center_in, 7);
	cudaMalloc(&center_out, 1);
	cudaMalloc(&sides_in, 4);
	cudaMalloc(&sides_out, 8);
	cudaMalloc(&corners_in, 4);
	cudaMalloc(&corners_out, 4);
*/	hipMalloc(&d1_in, size);
	hipMalloc(&d2_in, size);
	hipMalloc(&d_out, size);
	hipMalloc(&d_in, size);
	
//	for(int i=0; i<iterations; i++){
	hipMemcpy(d1_in, u1, size, hipMemcpyHostToDevice);
	hipMemcpy(d2_in, u2, size, hipMemcpyHostToDevice);

	updateCenter<<<1, size>>>(d_out, d1_in, d2_in);

	hipMemcpy(u, d_out, size, hipMemcpyDeviceToHost);	

	hipMemcpy(d_in, u, size, hipMemcpyHostToDevice);

	updateSides<<<1, size>>>(d_out, d_in);

	hipMemcpy(u, d_out, size, hipMemcpyDeviceToHost);

	hipMemcpy(d_in, u, size, hipMemcpyHostToDevice);

	updateCorners<<<1, size>>>(d_out, d_in);

	hipMemcpy(u, d_out, size, hipMemcpyDeviceToHost);
	
	for(int j=0; j<size; j++){
		u2[j] = u1[j];
	}

	for(int j=0; j<size; j++){
		u1[j] = u[j];
	}
	
	for(int j=0; j<size; j++){
		printf("%f, ", u[j]);
	}
	printf("\n");
//	}
}
