#include "hip/hip_runtime.h"
#include "all_frames.h"
#include <cuPrintf.cuh>
#include <cuPrintf.cu>
#include <stdio.h>
#include <stdlib.h>

#define BOX_WIDTH 16
#define SEARCH_BOUNDARY 7

__global__ void sum(float * d_x, float * d_y, float * dans, int num_frames) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
//printf("thread%d, x %f, y %f", idx, d_x[idx],d_y[idx]);
	dans[0] += d_x[idx];
	dans[1] += d_y[idx];
	
	__syncthreads();
}

// Use exhaustive search Block Matching Motion Estimation algorithm
__global__ void estimate(float * d_x, float * d_y, int num_frames, unsigned char * d_frames) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned char* image1 = &d_frames[idx+1];
  unsigned char* image2 = &d_frames[idx];
  unsigned int  x2, y2, box_count;
  int width = 480, height = 360;
  float total_x, total_y;
  int m, n, dy, dx, x1, y1, min_cost, curr_cost;
  box_count = 1;
//printf ("Thread number %d. f = %d\n", threadIdx.x, idx);

  for (y2 = 0; y2 < height - BOX_WIDTH; y2 += BOX_WIDTH) {
    for (x2 = 0; x2 < width - BOX_WIDTH; x2 += BOX_WIDTH) {

      min_cost = 65537;
      dy = 0;
      dx = 0;

      for (m = -SEARCH_BOUNDARY; m < SEARCH_BOUNDARY; m++) {
        for (n = -SEARCH_BOUNDARY; n < SEARCH_BOUNDARY; n++) {
          x1 = x2 + m;
          y1 = y2 + n;
          if (x1 < 0 || y1 < 0 || x1 + BOX_WIDTH >= width ||
              y1 + BOX_WIDTH >= height) { // dont execute if out f bounds
            continue;
          }
          int i, j, m1, n1, m2, n2, diff;
			unsigned char im1, im2;
			unsigned int sum;
			sum = 0;
			
			for (i = 0; i < BOX_WIDTH; i++) {
				m1 = x1 + i;
				m2 = x2 + i;
				if (m1 < 0 || m2 < 0 || m1 >= height || m2 >= height) {
					curr_cost = 63557;
				}
				for (j = 0; j < BOX_WIDTH; j++) {

				n1 = y1 + j;
				n2 = y2 + j;
				if (n1 < 0 || n2 < 0 || n1 >= width || n2 >= width) {
					curr_cost = 63557;
				}
				im1 = image1[m1 + n1 * width];
				im2 = image2[m2 + width * n2];
				diff = im1 - im2;
				if (diff < 0) {
					diff = -diff;
				}
				sum += diff;
				}
			}
		  curr_cost = sum / (BOX_WIDTH * BOX_WIDTH);
          if (curr_cost < min_cost) { // calculate minimum cost
            min_cost = curr_cost;
            dx = m;
            dy = n;
          }
        }
      }
      if (min_cost >= 0 && min_cost < 65537) {
        total_y += dy;
        total_x += dx;
        box_count++;
        // printf("y2: %d, x2: %d, box: %d\n",y2,x2,box_count );
      }
    }
  }


  d_x[idx] = total_x / box_count; // other calculation can be done with this
  d_x[idx] = total_y / box_count;
}

int main(int argc, char **argv) {
  //unsigned char *frame_1, *frame_2;
  //int i;
  int width = 480, height = 360;
  const int FRAME_SIZE = sizeof(float) * (num_frames - 1);

  float *mean_x_array = (float *)malloc(sizeof(float) * (num_frames - 1));
  float *mean_y_array = (float *)malloc(sizeof(float) * (num_frames - 1)); 
  float *ans = (float *)malloc(sizeof(float) * 2);
  
  float *d_x;
  float *d_y;
  float *dans;
  unsigned char *d_frames;
 int numblocks =num_frames-1, numthreads=1; 
  
  hipMalloc(&d_x, FRAME_SIZE);
  hipMalloc(&d_y, FRAME_SIZE);
  hipMalloc(&dans, sizeof(float)*2);
  hipMalloc(&d_frames, sizeof(char)*width*height*100);
  printf("%f \n", num_frames);
  
  //for (i = 1; i < num_frames; i++) {
    //frame_1 = frames[i - 1];
    //frame_2 = frames[i];
    //estimate(frame_1, frame_2, &mean_x_array[i], &mean_y_array[i]);
  //}
  
  hipMemcpy(d_x, mean_x_array, FRAME_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_y, mean_y_array, FRAME_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_frames, frames, sizeof(char)*width*height*100, hipMemcpyHostToDevice);
  
  estimate<<<numblocks, numthreads>>>(d_x, d_y, num_frames, d_frames);
  {
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}
  
  
  hipMemcpy(dans, ans, sizeof(float)*2, hipMemcpyHostToDevice);
  
  sum<<<numblocks,numthreads>>>(d_x, d_y, dans, num_frames);
{
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",

               hipGetErrorString(cudaerr));
}
  
  hipMemcpy(dans, ans, sizeof(float)*2, hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
  
  printf("mean_x: %f, men_y %f\n", ans[0] / (num_frames - 1), ans[1] / (num_frames - 1));
  
  free(mean_x_array);
  free(mean_y_array);

  return 0;
}


